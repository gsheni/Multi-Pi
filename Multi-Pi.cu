#include "hip/hip_runtime.h"

// Gaurav Sheni
// CSC 391 
// December 2, 2015
// Project 4

#include <stdio.h>

__global__ void first_call();
__global__ void normalized_freq(char* digits, int* global_mem_freq_count, hipStream_t A);

//number of digits to read each time
#define number_of_digits 1000000

int main ( int argc, char *argv[] ) {

	hipSetDevice(0);
	first_call<<<1,1>>>();
	hipSetDevice(1);
	first_call<<<1,1>>>();

	clock_t start; // for starting
	clock_t stop; //for stoping
	double execution_time;	//total time 
	start = clock();	//ready set go

	///check for correct # of arguments
	if (argc != 2){
		printf ("Incorrect number of command line arugments.\r\n");
		exit(1);
	}

	FILE *file_read = fopen(argv[1], "r+");
	if (file_read == NULL) {
	    printf("File could not be read. ");
	    exit(1);
	}
	FILE *file_output = fopen("freq.dat",  "w+");
	if (file_output == NULL) {
	    printf("File, freq.dat, could not be created.");
	    exit(1);
	}

	int* A_freq_count;
	int* B_freq_count;
	char* A_digits;
	char* B_digits;
	int* A2_freq_count;
	int* B2_freq_count;
	char* A2_digits;
	char* B2_digits;
	char* input_string = (char*)malloc(sizeof(char) * number_of_digits);
	hipHostAlloc((void**) &A_digits, sizeof(char) * number_of_digits, hipHostMallocDefault);
	hipHostAlloc((void**) &B_digits, sizeof(char) * number_of_digits, hipHostMallocDefault);
	hipHostAlloc((void**) &A_freq_count, sizeof(int) * 10, hipHostMallocDefault);
	hipHostAlloc((void**) &B_freq_count, sizeof(int) * 10, hipHostMallocDefault);
	hipHostAlloc((void**) &A2_digits, sizeof(char) * number_of_digits, hipHostMallocDefault);
	hipHostAlloc((void**) &B2_digits, sizeof(char) * number_of_digits, hipHostMallocDefault);
	hipHostAlloc((void**) &A2_freq_count, sizeof(int) * 10, hipHostMallocDefault);
	hipHostAlloc((void**) &B2_freq_count, sizeof(int) * 10, hipHostMallocDefault);
	for (int j = 0; j < 10 ; j++){
		A_freq_count[j] = 0;
		B_freq_count[j] = 0;
		A2_freq_count[j] = 0;
		B2_freq_count[j] = 0;
	}

	int device_sync_count = 1;

	int *dev_A_freq_count;
	char *dev_A_digits;
	int *dev_B_freq_count;
	char *dev_B_digits;
	int *dev_A2_freq_count;
	char *dev_A2_digits;
	int *dev_B2_freq_count;
	char *dev_B2_digits;

	hipStream_t stream1;
	hipStreamCreate(&stream1);
	hipStream_t stream3;
	hipStreamCreate(&stream3);
	hipStream_t stream2;
	hipStreamCreate(&stream2);
	hipStream_t stream4;
	hipStreamCreate(&stream4);


	hipSetDevice(0);
	hipMalloc((void**) &dev_A_freq_count, 10*sizeof(int));
	hipMalloc((void**) &dev_A_digits, number_of_digits*sizeof(char));
	hipMemcpyAsync(dev_A_freq_count, A_freq_count, 10 * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMalloc((void**) &dev_A2_freq_count, 10*sizeof(int));
	hipMalloc((void**) &dev_A2_digits, number_of_digits*sizeof(char));
	hipMemcpyAsync(dev_A2_freq_count, A2_freq_count, 10 * sizeof(int), hipMemcpyHostToDevice, stream3);

	hipSetDevice(1);
	hipMalloc((void**) &dev_B_freq_count, 10*sizeof(int));
	hipMalloc((void**) &dev_B_digits, number_of_digits*sizeof(char));
	hipMemcpyAsync(dev_B_freq_count, B_freq_count, 10 * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMalloc((void**) &dev_B2_freq_count, 10*sizeof(int));
	hipMalloc((void**) &dev_B2_digits, number_of_digits*sizeof(char));
	hipMemcpyAsync(dev_B2_freq_count, B2_freq_count, 10 * sizeof(int), hipMemcpyHostToDevice, stream4);

	fgetc(file_read);

	while(fgets(A_digits, number_of_digits + 1, file_read) != NULL ) {

		hipSetDevice(0);
		hipMemcpyAsync(dev_A_digits, A_digits, number_of_digits * sizeof(char), hipMemcpyHostToDevice, stream1);
		fgets(A2_digits, number_of_digits + 1, file_read);
		hipMemcpyAsync(dev_A2_digits, A2_digits, number_of_digits * sizeof(char), hipMemcpyHostToDevice, stream3);
		hipSetDevice(1);
		fgets(B_digits, number_of_digits + 1, file_read);
		hipMemcpyAsync(dev_B_digits, B_digits, number_of_digits * sizeof(char), hipMemcpyHostToDevice, stream2);
		fgets(B2_digits, number_of_digits + 1, file_read);
		hipMemcpyAsync(dev_B2_digits, B2_digits, number_of_digits * sizeof(char), hipMemcpyHostToDevice, stream4);
		hipSetDevice(0);

		normalized_freq<<<(int)ceil(number_of_digits/235) + 1, 235>>>(dev_A_digits,  dev_A_freq_count, stream1);
		normalized_freq<<<(int)ceil(number_of_digits/235) + 1, 235>>>(dev_A2_digits,  dev_A2_freq_count, stream3);
		hipSetDevice(1);
		normalized_freq<<<(int)ceil(number_of_digits/235) + 1, 235>>>(dev_B_digits,  dev_B_freq_count, stream2);
		normalized_freq<<<(int)ceil(number_of_digits/235) + 1, 235>>>(dev_B2_digits,  dev_B2_freq_count, stream4);
		hipDeviceSynchronize();
		printf("GPUs Synchronized (%i)\n", device_sync_count);
		device_sync_count++;
	}

	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream2);
	hipStreamSynchronize(stream3);
	hipStreamSynchronize(stream4);
	hipSetDevice(0);
	hipMemcpyAsync(A_freq_count, dev_A_freq_count, 10 * sizeof(int), hipMemcpyDeviceToHost, stream1);
	hipMemcpyAsync(A2_freq_count, dev_A2_freq_count, 10 * sizeof(int), hipMemcpyDeviceToHost, stream3);
	hipSetDevice(1);
	hipMemcpyAsync(B_freq_count, dev_B_freq_count, 10 * sizeof(int), hipMemcpyDeviceToHost, stream2);
	hipMemcpyAsync(B2_freq_count, dev_B2_freq_count, 10 * sizeof(int), hipMemcpyDeviceToHost, stream4);
	hipFree(dev_A_freq_count);
	// hipFree(dev_A_digits);
	// hipFree(dev_B_freq_count);
	// hipFree(dev_B_digits);
	// hipFree(dev_A2_freq_count);
	// hipFree(dev_A2_digits);
	// hipFree(dev_B2_freq_count);
	// hipFree(dev_B2_digits);
	// hipStreamDestroy(stream1);
	// hipStreamDestroy(stream2);
	// hipStreamDestroy(stream3);
	// hipStreamDestroy(stream4);

	A_freq_count[3]++;
	int total = 0;
	for (int i = 0; i < 10; i++) {
		printf("Frequence at %i, is %i\n", i, A_freq_count[i] + B_freq_count[i]+ A2_freq_count[i] + B2_freq_count[i]);
		float output = (float) ( A_freq_count[i] + B_freq_count[i]+ A2_freq_count[i] + B2_freq_count[i] ) / (float) 100000001;

		fprintf(file_output,"%i\t%f\n", i, output);
		total = total + A_freq_count[i] + B_freq_count[i] + A2_freq_count[i] + B2_freq_count[i];
	}
	printf("Total Frequency: %i\n", total);
	//STOP
	stop = clock();

	//get the execution time
	execution_time = ((double) (stop - start)) / CLOCKS_PER_SEC;
	//Print the execution time

	printf("Execution Time in Seconds: %.8lf\n", execution_time );

	// fclose(file_output);
	// fclose(file_read);

	//exit the program, done
	exit(0);

}

//dummy function
__global__ void first_call(){
	int z = 1;
	if ( z != 1 ){
	}
}
__global__ void normalized_freq(char* digits, int* global_mem_freq_count, hipStream_t A) {

	int global_ID = blockDim.x * blockIdx.x + threadIdx.x;

	if (global_ID >= number_of_digits ){
		return;
	}
	

	__shared__ int block_freq_count[10];

	if (threadIdx.x == 0){
		memset(block_freq_count, 0, 10 * sizeof(int));
	}
	syncthreads();

	atomicAdd(&block_freq_count[digits[global_ID] - '0'], 1);

	syncthreads();
	if (threadIdx.x == 0){
		for (int i = 0; i< 10 ; i++){
			atomicAdd(&global_mem_freq_count[i], block_freq_count[i]);
		}
	}
}
void CUDAErrorCheck()
{
        hipError_t error = hipGetLastError();
        if (error != hipSuccess)
        {
                printf("CUDA -error : %s (%d)\n", hipGetErrorString(error), error);
                //exit(0);
        }
}